#include "hip/hip_runtime.h"
// Another massive improvement by leveraging on-chip shared memory
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "files.h"

#define SOFTENING 1e-9f

// Constants
#define BLOCK_SIZE 256

__global__ void bodyForceShared(float *x, float *y, float *z,
                                float *vx, float *vy, float *vz,
                                float dt, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= n) return;

    float Fx = 0.0f, Fy = 0.0f, Fz = 0.0f;

    float myX = x[i];
    float myY = y[i];
    float myZ = z[i];

    __shared__ float sh_x[BLOCK_SIZE];
    __shared__ float sh_y[BLOCK_SIZE];
    __shared__ float sh_z[BLOCK_SIZE];

    for (int tile = 0; tile < n; tile += BLOCK_SIZE) {
        int idx = tile + threadIdx.x;
        if (idx < n) {
            sh_x[threadIdx.x] = x[idx];
            sh_y[threadIdx.x] = y[idx];
            sh_z[threadIdx.x] = z[idx];
        } else {
            sh_x[threadIdx.x] = 0.0f;
            sh_y[threadIdx.x] = 0.0f;
            sh_z[threadIdx.x] = 0.0f;
        }
        __syncthreads();

        #pragma unroll
        for (int j = 0; j < BLOCK_SIZE; j++) {
            float dx = sh_x[j] - myX;
            float dy = sh_y[j] - myY;
            float dz = sh_z[j] - myZ;
            float distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            float invDist = rsqrtf(distSqr);
            float invDist3 = invDist * invDist * invDist;
            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }
        __syncthreads();
    }

    vx[i] += dt * Fx;
    vy[i] += dt * Fy;
    vz[i] += dt * Fz;
}

__global__ void posIntegrate(float *x, float *y, float *z,
                              float *vx, float *vy, float *vz,
                              float dt, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= n) return;

    x[i] += vx[i] * dt;
    y[i] += vy[i] * dt;
    z[i] += vz[i] * dt;
}

int main(const int argc, const char** argv) {
    int nBodies = 2 << 11;
    if (argc > 1) nBodies = 2 << atoi(argv[1]);

    const char * initialized_values;
    const char * solution_values;
    if (nBodies == 2 << 11) {
        initialized_values = "09-nbody/files/initialized_4096";
        solution_values = "09-nbody/files/solution_4096";
    } else {
        initialized_values = "09-nbody/files/initialized_65536";
        solution_values = "09-nbody/files/solution_65536";
    }
    if (argc > 2) initialized_values = argv[2];
    if (argc > 3) solution_values = argv[3];

    const float dt = 0.01f;
    const int nIters = 10;

    int bytes = nBodies * sizeof(float);
    float *x, *y, *z, *vx, *vy, *vz;

    hipHostMalloc(&x, bytes);
    hipHostMalloc(&y, bytes);
    hipHostMalloc(&z, bytes);
    hipHostMalloc(&vx, bytes);
    hipHostMalloc(&vy, bytes);
    hipHostMalloc(&vz, bytes);

    float *tmp = (float*)malloc(nBodies * 6 * sizeof(float));
    read_values_from_file(initialized_values, tmp, nBodies * 6 * sizeof(float));

    for (int i = 0; i < nBodies; i++) {
        x[i] = tmp[i * 6 + 0];
        y[i] = tmp[i * 6 + 1];
        z[i] = tmp[i * 6 + 2];
        vx[i] = tmp[i * 6 + 3];
        vy[i] = tmp[i * 6 + 4];
        vz[i] = tmp[i * 6 + 5];
    }
    free(tmp);

    float *d_x, *d_y, *d_z, *d_vx, *d_vy, *d_vz;
    hipMalloc(&d_x, bytes);
    hipMalloc(&d_y, bytes);
    hipMalloc(&d_z, bytes);
    hipMalloc(&d_vx, bytes);
    hipMalloc(&d_vy, bytes);
    hipMalloc(&d_vz, bytes);

    hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_z, z, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vx, vx, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vy, vy, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_vz, vz, bytes, hipMemcpyHostToDevice);

    int numBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;

    double totalTime = 0.0;
    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();

        bodyForceShared<<<numBlocks, BLOCK_SIZE>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, dt, nBodies);
        posIntegrate<<<numBlocks, BLOCK_SIZE>>>(d_x, d_y, d_z, d_vx, d_vy, d_vz, dt, nBodies);

        hipDeviceSynchronize();
        totalTime += GetTimer() / 1000.0;
    }

    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / (totalTime / nIters);
    printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

    hipMemcpy(x, d_x, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(z, d_z, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(vx, d_vx, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(vy, d_vy, bytes, hipMemcpyDeviceToHost);
    hipMemcpy(vz, d_vz, bytes, hipMemcpyDeviceToHost);

    float *result = (float*)malloc(nBodies * 6 * sizeof(float));
    for (int i = 0; i < nBodies; i++) {
        result[i * 6 + 0] = x[i];
        result[i * 6 + 1] = y[i];
        result[i * 6 + 2] = z[i];
        result[i * 6 + 3] = vx[i];
        result[i * 6 + 4] = vy[i];
        result[i * 6 + 5] = vz[i];
    }
    write_values_to_file(solution_values, result, nBodies * 6 * sizeof(float));
    free(result);

    hipFree(d_x); hipFree(d_y); hipFree(d_z);
    hipFree(d_vx); hipFree(d_vy); hipFree(d_vz);
    hipHostFree(x); hipHostFree(y); hipHostFree(z);
    hipHostFree(vx); hipHostFree(vy); hipHostFree(vz);

    return 0;
}
